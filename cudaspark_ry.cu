#include "hip/hip_runtime.h"

//CUDA sticky2011 test --  Ananya Pavuluri

/* Monte - Carlo model of Ca spark from single cluster of RyRs
Parameters specified in: Ramay et. al, Cardiovascular Research, 2011
Model from Sobie et. al, Biophysics Journal, 2002 */

#include <iostream>
#include <cmath>
#include <iomanip>
#include <cstdlib>
#include <stdio.h>
#include <fstream>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

#define trials 100
using namespace std; // NOTE: When the number of trials is changed, the number of blocks in kernel launch changes as well.

__global__ void simulation(double F, double V_ss, double V_JSR, double tau_efflux, double tau_refill, double D_ryr, double EJequiv,
	double kr_minus, double kr_plus_max, double kr_plus_ryanodine, double Km_r_max, double Km_r_ryanodine, double alpha_r, int hill, int N_RyR, int N_RyR_ryanodine, double kcoup, double CSQ, double KCSQ,
    	double dt, const double dt_record, const double interval, const double timeafter, double t_end, int iterations, int outputs,
	double *devCassall, double *devCaJSRall, double *devIrelall, double *devNopenall, double *devNopenryanodine);


__global__ void simulation(double F, double V_ss, double V_JSR, double tau_efflux, double tau_refill, double D_ryr, double EJequiv,
	double kr_minus, double kr_plus_max, double kr_plus_ryanodine, double Km_r_max, double Km_r_ryanodine, double alpha_r, int hill, int N_RyR,
	int N_RyR_ryanodine, double kcoup, double CSQ, double KCSQ, double dt, const double dt_record, const double interval,
	const double timeafter, double t_end, int iterations, int outputs, double *devCassall, double *devCaJSRall, double *devIrelall, double *devNopenall, double *devNopenryanodine)
{
	// initializing CUDA random number generator
	hiprandState rndState;
	hiprand_init(clock64(), 13, 0, &rndState);

	// Subspace buffers 

	// Total [calmodulin], Total [SR membrane buffer],  Total [SL membrane buffer]
	double bt[3] = { 24, 47, 900 }; // uM
									// On rates for calmodulin, SR membrane, SL membrane
	double k_on[3] = { 100, 115, 115 };
	// Off rates for calmodulin, SR membrane, SL membrane
	double k_off[3] = { 38, 100, 1000 };
	// converting from s^-1 to ms^-1
	for (int n = 0; n < 3; n++) {
		k_on[n] *= 0.0010;
		k_off[n] *= 0.0010;
	}


	// Fixed ionic concentrations
	double Ca_myo = 0.1;        // bulk myoplasm [Ca2+]
	double Ca_NSR = 1000;		  // NSR [Ca2+]
	double Ca_ss = 0.1;         // subspace [Ca2+]
	double Ca_JSR = 1000;       // JSR [Ca2+]

								// Making array for buffers
	double b[3];

	double J_ryr = 0;
	double kr_plus_max_ryanodine = 23052865.2;
	// ---------------------        SIMULATION       --------------------- // 

	// INITIAL CONDITIONS
	Ca_ss = Ca_myo;
	Ca_JSR = Ca_NSR;
	double nopen_normal = 0;
	double nopen_ryanodine = 0;
	double nopen = nopen_normal + nopen_ryanodine;

	// buffers
	for (int i = 0; i < 3; i++) {
		b[i] = (bt[i] * (k_off[i] / k_on[i])) / (k_off[i] / k_on[i] + Ca_ss);
	}

	int writedex = 0;
	double tlast = -1 * dt;
	int J_d = 0;
	bool neverspark = true;
	double time = 0.0;
	
	//printf("before simulation");
	
	for (int j = 0; j < iterations; j++) {


		/*if (time >= interval && tlast < interval) {
			nopen = nopen + 5;
			printf("in\n");
		}
		/*else if (time >= interval + 10 && nopen < 1 && neverspark) {
			break;
		} */

		double nclosed = N_RyR - nopen;
		double nclosed_ryanodine = N_RyR_ryanodine - nopen_ryanodine;
		double nclosed_normal = (N_RyR - N_RyR_ryanodine) - nopen_normal;


		// Fluxes and currents	
		J_ryr = nopen * D_ryr * (Ca_JSR - Ca_ss) / V_ss; // uM/ms
		double I_ryr = 1e6 * J_ryr * 2 * F * V_ss; 	 // pA
		double J_efflux = (Ca_myo - Ca_ss) / tau_efflux;
		double J_refill = (Ca_NSR - Ca_JSR) / tau_refill;

		// Buffers
		double db_dt[3];
		double J_buff = 0;
		for (int i = 0; i < 3; i++) {
			db_dt[i] = -1 * k_on[i] * b[i] * Ca_ss + k_off[i] * (bt[i] - b[i]);
			J_buff += db_dt[i];
		}

		double denom = pow(KCSQ + Ca_JSR, 2);
		double B_JSR = pow((1 + CSQ * KCSQ / denom), -1);

		// Writing arrays after the fluxes are calculated, before integration
		// and state switching

		if (j % (iterations / (outputs - 1)) == 0) {
			dt = 0.0000100; // the value of dt changes to 9.99999974737875e-06 
							// within this conditional for an unknown reason.
							// here, it is assigned 1e-5 again to prevent this.
							//printf("block: %d\n", blockIdx.x);
			//printf("nopen: %lf\n", nopen);
			//printf("iteration: %d\n", j);
			int idx = blockDim.x * blockIdx.x + threadIdx.x;
			devIrelall[idx * outputs + writedex] = I_ryr;
			devNopenall[idx * outputs + writedex] = nopen;
			devNopenryanodine[idx * outputs + writedex] = nopen_ryanodine;
			devCassall[idx * outputs + writedex] = Ca_ss;
			devCaJSRall[idx * outputs + writedex] = Ca_JSR;
			writedex = writedex + 1;
		}

		double Km_r = Km_r_max - alpha_r * Ca_JSR;
		double pow1 = pow(Ca_ss, hill);
		double pow2 = pow(Km_r, hill);
		double kr_plus = kr_plus_max * pow1 / (pow1 + pow2);
		kr_plus_ryanodine = kr_plus_max_ryanodine * pow(0.1, hill) / (pow(0.1, hill) + pow(Km_r_ryanodine, hill));

		// Stochastic variables
		double  pincrease = dt * nclosed * kr_plus * pow(kcoup, 2 * nopen + 1 - N_RyR);
		//printf("pinc: %lf\n", pincrease);
		double  pdecrease = dt * nopen * kr_minus * pow(kcoup, 2 * nclosed + 1 - N_RyR);
		double	pincrease_ryanodine = dt*nclosed_ryanodine*kr_plus_ryanodine*pow(kcoup, (2 * nopen + 1 - N_RyR));
		//printf("pinc: %lf\n", pincrease_ryanodine);

		// opening of a free RyR
		if (hiprand_uniform(&rndState) < pincrease) {
			if ((nopen + 1) <= N_RyR) {
				nopen_normal = nopen_normal + 1;
			}
		}

		nopen += nopen_normal;

		// opening for ryanodine bound RyR
		if (hiprand_uniform(&rndState) < pincrease_ryanodine) {
			if ((nopen + 1) <= N_RyR) {
				nopen_ryanodine = nopen_ryanodine + 1;
			}
		}
		nopen += nopen_ryanodine;
		// closing free RyR
		if (nopen_normal != 0) {
			if (hiprand_uniform(&rndState) < pdecrease) {
				nopen_normal = nopen_normal - 1;
			}
		}
		// closing ryanodine-bound RyR
		if (nopen_ryanodine != 0) {
			if (hiprand_uniform(&rndState) < pdecrease) {
				nopen_ryanodine = nopen_ryanodine - 1;
			}
		}

		nopen = nopen_normal + nopen_ryanodine;

		if (nopen >= 5) {
			neverspark = false;
		}

		// Subspace [Ca2+] and JSR [Ca2+] derivatives
		double dCass_dt = J_efflux + J_d + J_ryr + J_buff;
		double dCaJSR_dt = B_JSR * (J_refill - J_ryr * V_ss / V_JSR);

		Ca_ss = Ca_ss + dt * dCass_dt;
		Ca_JSR = Ca_JSR + dt * dCaJSR_dt;

		// updating buffers
		for (int i = 0; i < 3; i++) {
			b[i] += dt*db_dt[i];
		}

		tlast = time;
		time += dt;
		// accounting for precision problems with floating point numbers.
		// ensures that time is accurate to 5 decimal places.
		time = round(time * 100000) / 100000;
		
	}

	// Write values at last time point t_end after the loop terminates

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	devCassall[index * outputs + writedex] = Ca_ss;
	devCaJSRall[index * outputs + writedex] = Ca_JSR;
	devIrelall[index * outputs + writedex] = 1e6 * J_ryr * 2 * F * V_ss;
	devNopenall[index * outputs + writedex] = nopen;
	devNopenryanodine[index * outputs + writedex] = nopen_ryanodine;

	printf("kernel end");
}




// ---------------HOST MEMORY ---------------
int main(void)

{

	// Faraday's Constant. Used to convert flux to current
	double F = 96.485; // C/mmol

					   // Geometrical parameters
	double V_ss = 1.0000e-12;
	double V_JSR = 1.6000e-12;

	// Time constants, in ms
	// time constant for ...
	double tau_efflux = 1.78e-3;
	// time constant for NSR to JSR refilling
	double tau_refill = 6.5;

	//Coupling energy between RyRs
	double EJequiv = 0.1;

	// RyR permeability constant
	double D_ryr = 2.2e-12;

	// RyR gating parameters
	double kr_minus, kr_plus_max, Km_r_max, alpha_r, hill, kr_plus_ryanodine, Km_r_ryanodine;
	kr_minus = 0.48;        // max close rate, ms^-1
	kr_plus_max = 30.0;         // max open rate, ms^-1
	kr_plus_ryanodine = 0;		// open rate when ryanodine is bound
	Km_r_max = 19.87;           // sensitivity of opening to subspace Ca, uM
	alpha_r = 1.0e-3;           // luminal dependence factor 
	Km_r_ryanodine = Km_r_max - alpha_r * 0.1;			// sensitivity of opening with ryanodine
	hill = 4;                   // exponent

	int N_RyR = 28; // # of RyR channels in a cluster
	int N_RyR_ryanodine = 1; // # of ryanodine-bound RyRs

							 // Coupling rate
	double kcoup = exp(2 * EJequiv / (N_RyR - 1));


	// JSR buffer calsequestrin
	double CSQ = 30e3; // uM  // total [CSQ]
	double KCSQ = 630; // uM  // Ca dissociation constant 

	// time steps
	double dt = 0.0000100;
	const double dt_record = 0.1000000;

	// Open single RyR at interval, run for 'timeafter' ms
	const double interval = 1;
	const double timeafter = 2500.0; // change this to change amount of data.  

									//Initializing arrays to hold results
	double t_end = interval + timeafter;

	int iterations = nearbyint(t_end / dt);
	int outputs = nearbyint(t_end / dt_record) + 1;

	int plottime_size = (int)outputs * dt_record * 10;

	// ALLOCATING HEAP MEMORY FOR DYNAMIC ARRAYS

	// times for the plot
	double *plottime = new double[plottime_size];
	double timepoint = 0;
	for (int i = 0; i < plottime_size; i++) {
		plottime[i] = timepoint;
		timepoint = round((timepoint + dt_record) * 10) / 10;
	}
	// "host" arrays are flattened 1d arrays that will temporarily store:
	// JSR [Ca2+], subspace [Ca2+], number of RyRs open, and currents, respectively
	double *hostCaJSRall = new double[outputs * trials];
	double *devCaJSRall = new double[outputs * trials];

	// testing hipMalloc()
	if (hipSuccess != hipMalloc((void**)&devCaJSRall, outputs * trials * sizeof(double))) {
		cout << "Malloc fail" << endl;
	}
	double *hostCassall = new double[outputs* trials];
	double *devCassall = new double[outputs * trials];
	hipMalloc((void**)&devCassall, outputs * trials * sizeof(double));
	double *hostNopenall = new double[outputs * trials];
	double *devNopenall = new double[outputs * trials];
	hipMalloc((void**)&devNopenall, outputs * trials * sizeof(double));
	double *hostIrelall = new double[outputs * trials];
	double *devIrelall = new double[outputs * trials];
	hipMalloc((void**)&devIrelall, outputs * trials * sizeof(double));
	double *hostNopenryanodine = new double[outputs * trials];
	double *devNopenryanodine = new double[outputs * trials];
	hipMalloc((void**)&devNopenryanodine, outputs * trials * sizeof(double));


	// testing hipMemcpy()
	if (hipSuccess != hipMemcpy(devCaJSRall, hostCaJSRall, outputs * trials * sizeof(double), hipMemcpyHostToDevice)) {
		cout << "memcpy fail" << endl;
	}
	hipMemcpy(devCassall, hostCassall, outputs * trials * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devNopenall, hostNopenall, outputs * trials * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devIrelall, hostIrelall, outputs * trials * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(devNopenryanodine, hostNopenryanodine, outputs * trials * sizeof(double), hipMemcpyHostToDevice);

	cout << "Starting simulation" << endl;
	cout << outputs * trials << endl;
	simulation << <trials, 1 >> > (F, V_ss, V_JSR, tau_efflux, tau_refill, D_ryr, EJequiv, kr_minus, kr_plus_max, kr_plus_ryanodine, Km_r_max, Km_r_ryanodine, alpha_r, hill, N_RyR, N_RyR_ryanodine, kcoup, CSQ, KCSQ,
		dt, dt_record, interval, timeafter, t_end, iterations, outputs, devCassall, devCaJSRall, devIrelall, devNopenall, devNopenryanodine);
	if (hipDeviceSynchronize() != hipSuccess) {
		cout << "sync fail" << endl;
	}
	cout << "End of simulation" << endl;
	hipMemcpy(hostCassall, devCassall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devCassall);
	hipMemcpy(hostCaJSRall, devCaJSRall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devCaJSRall);
	hipMemcpy(hostNopenall, devNopenall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devNopenall);
	hipMemcpy(hostIrelall, devIrelall, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devIrelall);
	hipMemcpy(hostNopenryanodine, devNopenryanodine, outputs * trials * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(devNopenryanodine);

	// storing information from flattened 1d arrays into 2d arrays

	// Array for [Ca2+] in the JSR
	double **Ca_JSR_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Ca_JSR_all[i] = new double[trials];
	}

	// Array for subspace [Ca2+]
	double **Ca_ss_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Ca_ss_all[i] = new double[trials];
	}

	// Array for current released from RyR
	double **Irel_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Irel_all[i] = new double[trials];
	}

	// Array for number of receptors open
	double **Nopen_all = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Nopen_all[i] = new double[trials];
	}

	// Array for number of receptors open with ryanodine
	double **Nopen_ryanodine = new double *[outputs];
	for (int i = 0; i < outputs; i++) {
		Nopen_ryanodine[i] = new double[trials];
	}

	for (int x = 0; x < outputs; x++) {
		for (int y = 0; y < trials; y++) {
			Ca_JSR_all[x][y] = 0;
			Ca_ss_all[x][y] = 0;
			Irel_all[x][y] = 0;
			Nopen_all[x][y] = 0;
			Nopen_ryanodine[x][y] = 0;

		}
	}

	int count = 0;
	for (int x = 0; x < trials; x++) {
		for (int y = 0; y < outputs; y++) {
			Ca_JSR_all[y][x] = hostCaJSRall[count];
			Ca_ss_all[y][x] = hostCassall[count];
			Nopen_all[y][x] = hostNopenall[count];
			Irel_all[y][x] = hostIrelall[count];
			Nopen_ryanodine[y][x] = hostNopenryanodine[count];
			count++;
		}
	}

	cout << "Begin writing output" << endl;

	ofstream Nopen;
	Nopen.open("N_open.csv");
	while (Nopen.is_open()) {
		for (int i = 0; i < outputs; i++) {
			Nopen << Nopen_all[i][0];
			for (int j = 1; j < trials; j++) {
				Nopen << "," << Nopen_all[i][j];
			}
			Nopen << endl;
		}
		break;
	}
	Nopen.close();
	ofstream Irel;
	Irel.open("Irel.csv");
	while (Irel.is_open()) {
		for (int i = 0; i < outputs; i++) {
			Irel << Irel_all[i][0];
			for (int j = 1; j < trials; j++) {
				Irel << "," << Irel_all[i][j];
			}
			Irel << endl;
		}
		break;
	}
	Irel.close();
	ofstream Cads;
	Cads.open("Ca_ss.csv");
	while (Cads.is_open()) {
		for (int i = 0; i < outputs; i++) {
			Cads << Ca_ss_all[i][0];
			for (int j = 1; j < trials; j++) {
				Cads << "," << Ca_ss_all[i][j];
			}
			Cads << endl;
		}
		break;
	}
	Cads.close();
	ofstream CaJSR;
	CaJSR.open("CaJSR.csv");
	while (CaJSR.is_open()) {
		for (int i = 0; i < outputs; i++) {
			CaJSR << Ca_JSR_all[i][0];
			for (int j = 1; j < trials; j++) {
				CaJSR << "," << Ca_JSR_all[i][j];
			}
			CaJSR << endl;
		}
		break;
	}
	ofstream Nopen_ry;
	Nopen_ry.open("Nopenryanodine.csv");
	while (Nopen_ry.is_open()) {
		for (int i = 0; i < outputs; i++) {
			Nopen_ry << Nopen_ryanodine[i][0];
			for (int j = 1; j < trials; j++) {
				Nopen_ry << "," << Nopen_ryanodine[i][j];
			}
			Nopen_ry << endl;
		}
		break;
	}
	Nopen_ry.close();

	ofstream plot_time;
	plot_time.open("plottime.csv");
	while (plot_time.is_open()) {
		for (int i = 0; i < outputs; i++) {
			plot_time << plottime[i];
			if (i != (outputs - 1))
				plot_time << ",";
		}
		break;
	}
	plot_time.close();
	// Recycling memory
	for (int k = 0; k < trials; k++) {
		delete[] Ca_ss_all[k];
		delete[] Ca_JSR_all[k];
		delete[] Nopen_all[k];
		delete[] Nopen_ryanodine[k];
		delete[] Irel_all[k];
	}

	delete[] hostCaJSRall;
	delete[] hostCassall;
	delete[] hostNopenall;
	delete[] hostIrelall;
	delete[] hostNopenryanodine;

	cout << "End of program." << endl;

	return 0;
}

